#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include "bitmap_image.hpp"

using namespace std;

__global__ void color_to_grey(uchar3 *input_image, uchar3 *output_image, int width, int height)
{

    // TODO: Convert color to grayscale by mapping components of uchar3 to RGB
    // x -> R; y -> G; z -> B
    // Apply the formula:
    // output = 0.299f * R + 0.578f * G + 0.114f * B
    // Hint: First create a mapping from 2D block and grid locations to an
    // absolute 2D location in the image then use that to calculate a 1D offset
}


int main(int argc, char **argv)
{
    if (argc != 2) {
        cerr << "format: " << argv[0] << " { 24-bit BMP Image Filename }" << endl;
        exit(1);
    }
    
    bitmap_image bmp(argv[1]);

    if(!bmp)
    {
        cerr << "Image not found" << endl;
        exit(1);
    }

    int height = bmp.height();
    int width = bmp.width();
    
    cout << "Image dimensions:" << endl;
    cout << "height: " << height << " width: " << width << endl;

    cout << "Converting " << argv[1] << " from color to grayscale..." << endl;

    //Transform image into vector of doubles
    vector<uchar3> input_image;
    rgb_t color;
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            bmp.get_pixel(x, y, color);
            input_image.push_back( {color.red, color.green, color.blue} );
        }
    }

    vector<uchar3> output_image(input_image.size());

    uchar3 *d_in, *d_out;
    int img_size = (input_image.size() * sizeof(char) * 3);
    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);

    hipMemcpy(d_in, input_image.data(), img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, input_image.data(), img_size, hipMemcpyHostToDevice);

    // TODO: Fill in the correnct blockSize and gridSize
    // currently only one block with one thread is being launched
    dim3 dimGrid(ceil(1, 1, 1);
    dim3 dimBlock(1, 1, 1);

    color_to_grey<<< dimGrid , dimBlock >>> (d_in, d_out, width, height);
    hipDeviceSynchronize();

    hipMemcpy(output_image.data(), d_out, img_size, hipMemcpyDeviceToHost);
    
    
    //Set updated pixels
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            int pos = x * width + y;
            bmp.set_pixel(x, y, output_image[pos].x, output_image[pos].y, output_image[pos].z);
        }
    }

    cout << "Conversion complete." << endl;
    
    bmp.save_image("./grayscaled.bmp");

    hipFree(d_in);
    hipFree(d_out);
}